
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib> // For standard C library functions

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d,%d] From device\n", threadIdx.x, blockIdx.x); 
}

int main(void) { 
    std::cout << "Hello World from host!\n"; 
    print_from_gpu<<<10,10>>>();
    hipDeviceSynchronize();
    return 0; 
}